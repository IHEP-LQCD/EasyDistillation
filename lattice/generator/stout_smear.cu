#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>

#define get_x(_coord, _X) \
  (((_coord[3] * _X[2] + _coord[2]) * _X[1] + _coord[1]) * _X[0] + _coord[0])

template <typename T, int Nc>
class Matrix {
private:
  T data[Nc][Nc];

public:
  __device__ __host__ Matrix()
  {
    for (int i = 0; i < Nc; ++i) {
      for (int j = 0; j < Nc; ++j) {
        data[i][j] = 0;
      }
    }
  }

  __device__ __host__ Matrix(const T *source)
  {
    for (int i = 0; i < Nc; ++i) {
      for (int j = 0; j < Nc; ++j) {
        data[i][j] = source[i * Nc + j];
      }
    }
  }

  __device__ __host__ Matrix(const Matrix<T, Nc> &matrix)
  {
    for (int i = 0; i < Nc; ++i) {
      for (int j = 0; j < Nc; ++j) {
        data[i][j] = matrix[i][j];
      }
    }
  }

  __device__ __host__ const T *operator[](const int i) const
  {
    return data[i];
  }

  __device__ __host__ T *operator[](const int i)
  {
    return data[i];
  }

  __device__ __host__ void operator=(const Matrix<T, Nc> &rhs)
  {
    for (int i = 0; i < Nc; ++i) {
      for (int j = 0; j < Nc; ++j) {
        data[i][j] = rhs[i][j];
      }
    }
  }

  __device__ __host__ Matrix<T, Nc> operator+(const T &rhs)
  {
    Matrix<T, Nc> result;
    for (int i = 0; i < Nc; ++i) {
      for (int j = 0; j < Nc; ++j) {
        result[i][j] = data[i][j];
      }
      result[i][i] += rhs;
    }
    return result;
  }

  __device__ __host__ Matrix<T, Nc> operator+(const Matrix<T, Nc> &rhs)
  {
    Matrix<T, Nc> result;
    for (int i = 0; i < Nc; ++i) {
      for (int j = 0; j < Nc; ++j) {
        result[i][j] = data[i][j] + rhs[i][j];
      }
    }
    return result;
  }

  __device__ __host__ void operator+=(const Matrix<T, Nc> &rhs)
  {
    for (int i = 0; i < Nc; ++i) {
      for (int j = 0; j < Nc; ++j) {
        data[i][j] += rhs[i][j];
      }
    }
  }

  __device__ __host__ void operator-=(const T &rhs)
  {
    for (int i = 0; i < Nc; ++i) {
      data[i][i] -= rhs;
    }
  }

  __device__ __host__ Matrix<T, Nc> operator*(const T &rhs)
  {
    Matrix<T, Nc> result;
    for (int i = 0; i < Nc; ++i) {
      for (int j = 0; j < Nc; ++j) {
        result[i][j] = data[i][j] * rhs;
      }
    }
    return result;
  }

  __device__ __host__ Matrix<T, Nc> operator*(const Matrix<T, Nc> &rhs)
  {
    Matrix<T, Nc> result;
    for (int i = 0; i < Nc; ++i) {
      for (int j = 0; j < Nc; ++j) {
        result[i][j] = 0;
        for (int k = 0; k < Nc; ++k) {
          result[i][j] += data[i][k] * rhs[k][j];
        }
      }
    }
    return result;
  }
};

template <typename T, int Nc>
__device__ __host__ T trace(const Matrix<T, Nc> &matrix)
{
  T result = 0;
  for (int i = 0; i < Nc; ++i) {
    result += matrix[i][i];
  }
  return result;
}

template <typename T, int Nc>
__device__ __host__ Matrix<T, Nc> adjoint(const Matrix<T, Nc> &matrix)
{
  Matrix<T, Nc> result;
  for (int i = 0; i < Nc; ++i) {
    for (int j = 0; j < Nc; ++j) {
      result[i][j] = conj(matrix[j][i]);
    }
  }
  return result;
}

template <typename T, int Nc>
__device__ __host__ Matrix<T, Nc> antiherm(const Matrix<T, Nc> &matrix)
{
  Matrix<T, Nc> result;
  for (int i = 0; i < Nc; ++i) {
    for (int j = 0; j < Nc; ++j) {
      result[i][j].real((matrix[j][i].imag() + matrix[i][j].imag()) / 2.);
      result[i][j].imag((matrix[j][i].real() - matrix[i][j].real()) / 2.);
    }
  }
  result -= trace(result) / T(Nc);
  return result;
}

template <typename T>
__global__ void stout_smear(complex<T> *U_out, const complex<T> *U_in, const T rho, const int Lx, const int Ly, const int Lz, const int Lt)
{
  const int Nd = 4;
  const int Nc = 3;
  typedef Matrix<complex<T>, Nc> ColorMatrix;

  const int volume = Lx * Ly * Lz * Lt;
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int mu = blockIdx.y * blockDim.y + threadIdx.y;
  if (mu >= Nd - 1 || x >= volume) {
    return;
  }
  const int X[Nd] = {Lx, Ly, Lz, Lt};
  int coord[Nd] = {x % Lx, x / Lx % Ly, x / (Lx * Ly) % Lz, x / (Lx * Ly * Lz) % Lt};

  ColorMatrix U(U_in + (mu * volume + x) * Nc * Nc);
  ColorMatrix Q;
  for (int nu = 0; nu < Nd - 1; ++nu) {
    if (nu != mu) {
      ColorMatrix staple1(U_in + (nu * volume + get_x(coord, X)) * Nc * Nc);
      coord[nu] = (coord[nu] + 1 + X[nu]) % X[nu];
      staple1 = staple1 * ColorMatrix(U_in + (mu * volume + get_x(coord, X)) * Nc * Nc);
      coord[mu] = (coord[mu] + 1 + X[mu]) % X[mu];
      coord[nu] = (coord[nu] - 1 + X[nu]) % X[nu];
      staple1 = staple1 * adjoint(ColorMatrix(U_in + (nu * volume + get_x(coord, X)) * Nc * Nc));
      coord[mu] = (coord[mu] - 1 + X[mu]) % X[mu];

      coord[nu] = (coord[nu] - 1 + X[nu]) % X[nu];
      ColorMatrix staple2(U_in + (nu * volume + get_x(coord, X)) * Nc * Nc);
      staple2 = adjoint(staple2) * ColorMatrix(U_in + (mu * volume + get_x(coord, X)) * Nc * Nc);
      coord[mu] = (coord[mu] + 1 + X[mu]) % X[mu];
      staple2 = staple2 * ColorMatrix(U_in + (nu * volume + get_x(coord, X)) * Nc * Nc);
      coord[nu] = (coord[nu] + 1 + X[nu]) % X[nu];
      coord[mu] = (coord[mu] - 1 + X[mu]) % X[mu];

      Q += staple1 + staple2;
    }
  }
  Q = antiherm((Q * rho) * adjoint(U));

  ColorMatrix Q_sq = Q * Q;
  double c0 = trace(Q_sq * Q).real() / 3;
  double c1 = trace(Q_sq).real() / 2;
  double c0_max = 2 * c1 / 3 * sqrt(c1 / 3);
  int parity = 0;
  if (c0 < 0) {
    parity = 1;
    c0 *= -1;
  }
  double theta = acos(c0 / c0_max);
  double u = sqrt(c1 / 3) * cos(theta / 3);
  double w = sqrt(c1) * sin(theta / 3);
  double u_sq = u * u;
  double w_sq = w * w;
  double e_iu_real = cos(u);
  double e_iu_imag = sin(u);
  double e_2iu_real = cos(2 * u);
  double e_2iu_imag = sin(2 * u);
  double cos_w = cos(w);
  double sinc_w = 1 - w_sq / 6 * (1 - w_sq / 20 * (1 - w_sq / 42 * (1 - w_sq / 72)));
  if (abs(w) > 0.05) {
    sinc_w = sin(w) / w;
  }
  double f_denom = 1 / (9 * u_sq - w_sq);
  double f0_real = ((u_sq - w_sq) * e_2iu_real + e_iu_real * 8 * u_sq * cos_w + e_iu_imag * 2 * u * (3 * u_sq + w_sq) * sinc_w) * f_denom;
  double f0_imag = ((u_sq - w_sq) * e_2iu_imag - e_iu_imag * 8 * u_sq * cos_w + e_iu_real * 2 * u * (3 * u_sq + w_sq) * sinc_w) * f_denom;
  double f1_real = (2 * u * e_2iu_real - e_iu_real * 2 * u * cos_w + e_iu_imag * (3 * u_sq - w_sq) * sinc_w) * f_denom;
  double f1_imag = (2 * u * e_2iu_imag + e_iu_imag * 2 * u * cos_w + e_iu_real * (3 * u_sq - w_sq) * sinc_w) * f_denom;
  double f2_real = (e_2iu_real - e_iu_real * cos_w - e_iu_imag * 3 * u * sinc_w) * f_denom;
  double f2_imag = (e_2iu_imag + e_iu_imag * cos_w - e_iu_real * 3 * u * sinc_w) * f_denom;
  if (parity) {
    f0_imag *= -1;
    f1_real *= -1;
    f2_imag *= -1;
  }
  complex<T> f0 = {T(f0_real), T(f0_imag)};
  complex<T> f1 = {T(f1_real), T(f1_imag)};
  complex<T> f2 = {T(f2_real), T(f2_imag)};
  ColorMatrix e_iQ = Q_sq * f2 + Q * f1 + f0;
  U = e_iQ * U;
  for (int i = 0; i < Nc; ++i) {
    for (int j = 0; j < Nc; ++j) {
      U_out[(mu * volume + x) * Nc * Nc + i * Nc + j] = U[i][j];
    }
  }
}
